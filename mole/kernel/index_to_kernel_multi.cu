#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// CUDA kernel for indexing pinned memory
// pin_memory to CUDA
template <typename T>
__global__ void index_to_cuda_kernel(
    const float* __restrict__ input,
    const int* __restrict__ indices,
    T* __restrict__ output,
    const int batch_size,
    const int feature_dim,
    const int input_stride) {
    
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int batch_idx = tid / (feature_dim / _k_);

    if (batch_idx < batch_size)
    {
        int feat_idx = tid % (feature_dim / _k_) * _k_;
        int output_idx = tid * _k_;
        int start_idx = indices[batch_idx] * input_stride + feat_idx;
        #pragma unroll
        for (int i=0; i<_k_; i++)
            output[output_idx + i] = T(input[start_idx + i]);
    }
}

void index_to_cuda(
    const torch::Tensor& input,
    const torch::Tensor& indices,
    const torch::Tensor& output) {
    
    // Get dimensions
    int batch_size = indices.size(0);
    int feature_dim = input.size(1);
    int input_stride = input.stride(0);
    
    
    // Calculate grid and block sizes
    const int threads_per_block = 256;
    int total_threads = batch_size * feature_dim / _k_;
    int blocks = (total_threads + threads_per_block - 1) / threads_per_block;
    
    // Launch kernel
    if (output.dtype() == torch::kFloat)
        index_to_cuda_kernel<<<blocks, threads_per_block>>>(
            input.data_ptr<float>(),
            indices.data_ptr<int>(),
            output.data_ptr<float>(),
            batch_size,
            feature_dim,
            input_stride
        );
    else if (output.dtype() == torch::kHalf)
        index_to_cuda_kernel<<<blocks, threads_per_block>>>(
            input.data_ptr<float>(),
            indices.data_ptr<int>(),
            output.data_ptr<at::Half>(),
            batch_size,
            feature_dim,
            input_stride
        );
    else if (output.dtype() == torch::kBFloat16)
        index_to_cuda_kernel<<<blocks, threads_per_block>>>(
            input.data_ptr<float>(),
            indices.data_ptr<int>(),
            output.data_ptr<at::BFloat16>(),
            batch_size,
            feature_dim,
            input_stride
        );
    else
        throw std::runtime_error("Unsupported output dtype");
}
// CUDA to pin_memory
__global__ void index_to_pinned_kernel(
    float* __restrict__ input,
    const int* __restrict__ indices,
    const float* __restrict__ x,
    const int batch_size,
    const int feature_dim,
    const int input_stride) {
    
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (tid < batch_size * feature_dim) {
        int batch_idx = tid / feature_dim;
        int feat_idx = tid % feature_dim;
        
        int input_idx = indices[batch_idx];
        input[input_idx * input_stride + feat_idx] = x[tid];
    }
}
// input[indices] = x
void index_to_pinned(
    const torch::Tensor& input,
    const torch::Tensor& indices,
    const torch::Tensor& x) {
    
    // Get dimensions
    int batch_size = indices.size(0);
    int feature_dim = input.size(1);
    int input_stride = input.stride(0);
        
    // Calculate grid and block sizes
    const int threads_per_block = 256;
    const int blocks = (batch_size * feature_dim + threads_per_block - 1) / threads_per_block;
    
    // Launch kernel
    index_to_pinned_kernel<<<blocks, threads_per_block>>>(
        input.data_ptr<float>(),
        indices.data_ptr<int>(),
        x.data_ptr<float>(),
        batch_size,
        feature_dim,
        input_stride
    );
}